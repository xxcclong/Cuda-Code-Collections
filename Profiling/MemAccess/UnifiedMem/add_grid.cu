
#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>
#include <stdio.h>
#include <math.h>
 
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
      if (result != cudaSuccess) {
              fprintf(stderr, "CUDA Runtime Error: %s\n",
                                  cudaGetErrorString(result));
                  assert(result == cudaSuccess);
                    }
#endif
        return result;
}
// CUDA kernel to add elements of two arrays
__global__
void add(int n, float *x, float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}
 
int main(void)
{
    int N = 1<<20;
    float *x, *y;

    // Allocate Unified Memory -- accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Launch kernel on 1M elements on the GPU
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    
    hipEvent_t startEvent, endEvent;
    checkCuda( hipEventCreate(&startEvent));
    checkCuda( hipEventCreate(&endEvent));
    checkCuda( hipEventRecord(startEvent, 0));

    add<<<numBlocks, blockSize>>>(N, x, y);

    checkCuda(hipEventRecord(endEvent, 0));
    checkCuda(hipEventSynchronize(endEvent));
    float ti;
    checkCuda(hipEventElapsedTime(&ti, startEvent, endEvent));
    printf("used time %f\n", ti);
    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y) ;

    return 0;
}
